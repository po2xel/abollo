#include "hip/hip_runtime.h"
#include "Market/Model/DataAnalyzerImpl.h"

#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include "Market/Model/TradeDate.h"



namespace abollo
{



void DataAnalyzerImpl::LoadIndex(const std::string& /*aCode*/, const date::year_month_day& /*aStartDate*/, const date::year_month_day& /*aEndDate*/)
{
    // using soci::into;
    // using soci::use;

    // Price lPrice{};

    // mIndexDailyStmt.exchange(use(aCode, "code"));
    // mIndexDailyStmt.exchange(use(aStartDate, "start"));
    // mIndexDailyStmt.exchange(use(aEndDate, "end"));
    // mIndexDailyStmt.exchange(into(lPrice));

    // mIndexDailyStmt.define_and_bind();
    // mIndexDailyStmt.execute();

    // thrust::host_vector<float> lPriceData{DEFAULT_BUFFER_CAPACITY};

    // PagedMarketingTable<float, 10, low_tag, high_tag> lPagedTable;

    // while (mIndexDailyStmt.fetch())
    //{
    //    mDateBuffer.push_back(lPrice.date);

    //    lPriceData[mPriceCount]                               = lPrice.open;
    //    lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE]     = lPrice.close;
    //    lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 2] = lPrice.low;
    //    lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 3] = lPrice.high;
    //    lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 4] = lPrice.volume;
    //    lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 5] = lPrice.amount;

    //    // lPagedTable.push_back<low_tag>(lPrice.low);
    //    // lPagedTable.push_back<high_tag>(lPrice.high);

    //    lPagedTable.push_back(lPrice);

    //    ++mPriceCount;
    //}

    // mIndexDailyStmt.bind_clean_up();

    // std::copy(lPagedTable.begin<low_tag>(), lPagedTable.begin<low_tag>() + 10, std::ostream_iterator<float>(std::cout, "\t"));
    // std::cout << std::endl;

    // std::copy(lPagedTable.begin<high_tag>(), lPagedTable.begin<high_tag>() + 10, std::ostream_iterator<float>(std::cout, "\t"));
    // std::cout << std::endl;

    // mMarketingTable.Append<date_tag>(mDateBuffer.begin(), mDateBuffer.end());
    // mMarketingTable.Append<low_tag>(lPagedTable.begin<low_tag>(), lPagedTable.end<low_tag>());
    // mMarketingTable.Append<high_tag>(lPagedTable.begin<high_tag>(), lPagedTable.end<high_tag>());

    // std::copy(mMarketingTable.begin<date_tag>(), mMarketingTable.begin<date_tag>() + 10, std::ostream_iterator<date::year_month_day>(std::cout, "\t"));
    // std::cout << std::endl;

    // std::copy(mMarketingTable.begin<low_tag>(), mMarketingTable.begin<low_tag>() + 10, std::ostream_iterator<float>(std::cout, "\t"));
    // std::cout << std::endl;

    // std::copy(mMarketingTable.begin<high_tag>(), mMarketingTable.begin<high_tag>() + 10, std::ostream_iterator<float>(std::cout, "\t"));

    // mDeviceBuffer = lPriceData;
}


std::tuple<float, float, float, float> DataAnalyzerImpl::MinMax(const std::size_t aStartIndex, const std::size_t aSize) const
{
    if (aStartIndex > mPriceCount)
        throw std::out_of_range("Start index out of range.");

    const auto lEndIndex = aStartIndex + std::min(aSize, mPriceCount - aStartIndex);

    const auto lMinIter = thrust::min_element(mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + lEndIndex);
    const auto lMaxIter = thrust::max_element(mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + lEndIndex);

    const auto lVolMinMaxIter =
        thrust::minmax_element(mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + lEndIndex);

    return std::make_tuple(*lMinIter, *lMaxIter, *lVolMinMaxIter.first, *lVolMinMaxIter.second);
}


std::pair<float, float> DataAnalyzerImpl::MinMax(const std::size_t aStartIndex, const std::size_t aSize, const ColumnTraits<price_tag>) const
{
    if (aStartIndex > mPriceCount)
        throw std::out_of_range("Start index out of range.");

    const auto lEndIndex = aStartIndex + std::min(aSize, mPriceCount - aStartIndex);

    const auto lMinIter = thrust::min_element(mMarketingTable.begin<low_tag>() + aStartIndex, mMarketingTable.begin<low_tag>() + lEndIndex);
    const auto lMaxIter = thrust::min_element(mMarketingTable.begin<high_tag>() + aStartIndex, mMarketingTable.begin<high_tag>() + lEndIndex);

    return std::make_pair(*lMinIter, *lMaxIter);
}


std::pair<DatePriceZipIterator, DatePriceZipIterator> DataAnalyzerImpl::Saxpy(const std::size_t aStartIndex, const std::size_t aSize, const float aScaleX, const float aTransX,
                                                                              const float aScaleY, const float aTransY, const float aScaleZ, const float aTransZ) const
{
    using thrust::placeholders::_1;

    if (aStartIndex > mPriceCount)
        throw std::out_of_range("Start index out of range.");

    const auto lSize     = std::min(aSize, mPriceCount - aStartIndex);
    const auto lEndIndex = aStartIndex + lSize;

    const thrust::counting_iterator<int> lCounterBegin{static_cast<int>(aStartIndex)};
    const thrust::counting_iterator<int> lCounterEnd{static_cast<int>(aStartIndex + lSize)};

    const auto lBeginIter = thrust::make_zip_iterator(
        make_tuple(lCounterBegin, mDeviceBuffer.begin() + DEFAULT_BUFFER_OPEN_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_CLOSE_POS + aStartIndex,
                   mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + aStartIndex,
                   mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_AMOUNT_POS + aStartIndex));

    const auto lEndIter =
        thrust::make_zip_iterator(make_tuple(lCounterEnd, mDeviceBuffer.begin() + DEFAULT_BUFFER_OPEN_POS + lEndIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_CLOSE_POS + lEndIndex,
                                             mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + lEndIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + lEndIndex,
                                             mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + lEndIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_AMOUNT_POS + lEndIndex));

    mDeviceTempBuffer.clear();

    thrust::transform(lBeginIter, lEndIter, mDeviceTempBuffer.begin(),
                      [sx = aScaleX, tx = aTransX, sy = aScaleY, ty = aTransY, sz = aScaleZ, tz = aTransZ] __device__(const IntFloat6& a) {
                          return thrust::make_tuple(a.get<0>() * sx + tx, a.get<1>() * sy + ty, a.get<2>() * sy + ty, a.get<3>() * sy + ty, a.get<4>() * sy + ty,
                                                    a.get<5>() * sz + tz, a.get<6>() * sz + tz);
                      });

    mHostTempBuffer.assign(mDeviceTempBuffer.begin(), mDeviceTempBuffer.begin() + lSize);

    // return std::make_pair(thrust::make_zip_iterator(thrust::make_tuple(mDateBuffer.begin() + aStartIndex, mHostTempBuffer.begin())),
    //                       thrust::make_zip_iterator(thrust::make_tuple(mDateBuffer.begin() + lEndIndex, mHostTempBuffer.end())));

    return std::make_pair(thrust::make_zip_iterator(thrust::make_tuple(mMarketingTable.begin<date_tag>() + aStartIndex, mHostTempBuffer.begin())),
                          thrust::make_zip_iterator(thrust::make_tuple(mMarketingTable.begin<date_tag>() + lEndIndex, mHostTempBuffer.end())));
}



}    // namespace abollo
