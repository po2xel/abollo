#include "hip/hip_runtime.h"
#include "Market/Model/IndexImpl.h"

#include "Market/Model/TradeDate.h"


#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>


namespace abollo
{



void IndexImpl::LoadIndex(const date::year_month_day& aStartDate, const date::year_month_day& aEndDate)
{
    using soci::into;
    using soci::use;

    Price lPrice{};

    mIndexDailyStmt.exchange(use(aStartDate, "start"));
    mIndexDailyStmt.exchange(use(aEndDate, "end"));
    mIndexDailyStmt.exchange(into(lPrice));

    mIndexDailyStmt.define_and_bind();
    mIndexDailyStmt.execute();

    thrust::host_vector<float> lPriceData{DEFAULT_BUFFER_CAPACITY};

    while (mIndexDailyStmt.fetch())
    {
        mDateBuffer.push_back(lPrice.date);

        lPriceData[mPriceCount]                               = lPrice.open;
        lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE]     = lPrice.close;
        lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 2] = lPrice.low;
        lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 3] = lPrice.high;
        lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 4] = lPrice.volume;
        lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 5] = lPrice.amount;

        ++mPriceCount;
    }

    mIndexDailyStmt.bind_clean_up();

    mDeviceBuffer = lPriceData;
}


std::tuple<float, float, float, float> IndexImpl::MinMax(const std::size_t aStartIndex, const std::size_t aSize) const
{
    if (aStartIndex > mPriceCount)
        throw std::out_of_range("Start index out of range.");

    const auto lEndIndex = aStartIndex + std::min(aSize, mPriceCount - aStartIndex);

    const auto lMinIter = thrust::min_element(mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + lEndIndex);
    const auto lMaxIter = thrust::max_element(mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + lEndIndex);

    const auto lVolMinMaxIter =
        thrust::minmax_element(mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + lEndIndex);

    return std::make_tuple(*lMinIter, *lMaxIter, *lVolMinMaxIter.first, *lVolMinMaxIter.second);
}


std::pair<DatePriceZipIterator, DatePriceZipIterator> IndexImpl::Saxpy(const std::size_t aStartIndex, const std::size_t aSize, const float aScaleX, const float aTransX,
                                                                       const float aScaleY, const float aTransY, const float aScaleZ, const float aTransZ) const
{
    using thrust::placeholders::_1;

    if (aStartIndex > mPriceCount)
        throw std::out_of_range("Start index out of range.");

    const auto lSize     = std::min(aSize, mPriceCount - aStartIndex);
    const auto lEndIndex = aStartIndex + lSize;

    const thrust::counting_iterator<int> lCounterBegin{static_cast<int>(aStartIndex)};
    const thrust::counting_iterator<int> lCounterEnd{static_cast<int>(aStartIndex + lSize)};

    const auto lBeginIter = thrust::make_zip_iterator(
        make_tuple(lCounterBegin, mDeviceBuffer.begin() + DEFAULT_BUFFER_OPEN_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_CLOSE_POS + aStartIndex,
                   mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + aStartIndex,
                   mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_AMOUNT_POS + aStartIndex));

    const auto lEndIter =
        thrust::make_zip_iterator(make_tuple(lCounterEnd, mDeviceBuffer.begin() + DEFAULT_BUFFER_OPEN_POS + lEndIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_CLOSE_POS + lEndIndex,
                                             mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + lEndIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + lEndIndex,
                                             mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + lEndIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_AMOUNT_POS + lEndIndex));

    mDeviceTempBuffer.clear();

    thrust::transform(lBeginIter, lEndIter, mDeviceTempBuffer.begin(),
                      [sx = aScaleX, tx = aTransX, sy = aScaleY, ty = aTransY, sz = aScaleZ, tz = aTransZ] __device__(const IntFloat6& a) {
                          return thrust::make_tuple(a.get<0>() * sx + tx, a.get<1>() * sy + ty, a.get<2>() * sy + ty, a.get<3>() * sy + ty, a.get<4>() * sy + ty,
                                                    a.get<5>() * sz + tz, a.get<6>() * sz + tz);
                      });

    mHostTempBuffer.assign(mDeviceTempBuffer.begin(), mDeviceTempBuffer.begin() + lSize);

    return std::make_pair(thrust::make_zip_iterator(thrust::make_tuple(mDateBuffer.begin() + aStartIndex, mHostTempBuffer.begin())),
                          thrust::make_zip_iterator(thrust::make_tuple(mDateBuffer.begin() + lEndIndex, mHostTempBuffer.end())));
}



}    // namespace abollo
