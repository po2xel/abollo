#include "hip/hip_runtime.h"
#include "Market/Model/IndexImpl.h"

#include "Market/Model/TradeDate.h"


#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>


namespace abollo
{



void IndexImpl::LoadMinMax(const date::year_month_day& aStartDate, const date::year_month_day& aEndDate)
{
    using soci::into;
    using soci::use;

    mMinMaxStmt.exchange(use(aStartDate, "start"));
    mMinMaxStmt.exchange(use(aEndDate, "end"));

    double lMinPrice{0}, lMaxPrice{0}, lMinVolume{0}, lMaxVolume{0}, lMinAmount{0}, lMaxAmount{0};
    mMinMaxStmt.exchange(into(lMinPrice));
    mMinMaxStmt.exchange(into(lMaxPrice));
    mMinMaxStmt.exchange(into(lMinVolume));
    mMinMaxStmt.exchange(into(lMaxVolume));
    mMinMaxStmt.exchange(into(lMinAmount));
    mMinMaxStmt.exchange(into(lMaxAmount));

    mMinMaxStmt.define_and_bind();
    mMinMaxStmt.execute(true);
    mMinMaxStmt.bind_clean_up();

    mMinMax = std::make_tuple(static_cast<float>(lMinPrice), static_cast<float>(lMaxPrice), static_cast<float>(lMinVolume / 1000000.f), static_cast<float>(lMaxVolume / 1000000.f),
                              static_cast<float>(lMinAmount / 1000000.f), static_cast<float>(lMaxAmount / 1000000.f));
}


void IndexImpl::LoadIndex(const date::year_month_day& aStartDate, const date::year_month_day& aEndDate)
{
    using soci::into;
    using soci::use;

    Price lPrice{};

    mIndexDailyStmt.exchange(use(aStartDate, "start"));
    mIndexDailyStmt.exchange(use(aEndDate, "end"));
    mIndexDailyStmt.exchange(into(lPrice));

    mIndexDailyStmt.define_and_bind();
    mIndexDailyStmt.execute();

    thrust::host_vector<float> lPriceData{DEFAULT_BUFFER_CAPACITY};

    while (mIndexDailyStmt.fetch())
    {
        mDateBuffer.push_back(lPrice.date);

        lPriceData[mPriceCount]                               = lPrice.open;
        lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE]     = lPrice.close;
        lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 2] = lPrice.low;
        lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 3] = lPrice.high;
        lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 4] = lPrice.volume;
        lPriceData[mPriceCount + DEFAULT_BUFFER_COL_SIZE * 5] = lPrice.amount;

        ++mPriceCount;
    }

    mIndexDailyStmt.bind_clean_up();

    mDeviceBuffer = lPriceData;
}


std::tuple<float, float, float, float> IndexImpl::MinMax(const std::size_t aStartIndex, const std::size_t aSize) const
{
    if (aStartIndex > mPriceCount)
        throw std::out_of_range("Start index out of range.");

    const auto lEndIndex = aStartIndex + std::min(aSize, mPriceCount - aStartIndex);

    const auto lMinIter = thrust::min_element(mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + lEndIndex);
    const auto lMaxIter = thrust::max_element(mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + lEndIndex);

    const auto lVolMinMaxIter =
        thrust::minmax_element(mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + lEndIndex);

    return std::make_tuple(*lMinIter, *lMaxIter, *lVolMinMaxIter.first, *lVolMinMaxIter.second);
}


std::tuple<float, float, float, float> IndexImpl::MinMax(const date::year_month_day& /*aStartDate*/, const date::year_month_day& /*aEndDate*/) const
{
    return std::make_tuple(1.f, .1f, 1.f, 1.f);
}


std::pair<DatePriceZipIterator, DatePriceZipIterator> IndexImpl::Saxpy(const std::size_t aStartIndex, const std::size_t aSize, const float aScaleX, const float aTransX,
                                                                       const float aScaleY, const float aTransY, const float aScaleZ, const float aTransZ) const
{
    using thrust::placeholders::_1;

    if (aStartIndex > mPriceCount)
        throw std::out_of_range("Start index out of range.");

    const auto lSize     = std::min(aSize, mPriceCount - aStartIndex);
    const auto lEndIndex = aStartIndex + lSize;

    const thrust::counting_iterator<int> lCounterBegin{static_cast<int>(aStartIndex)};
    const thrust::counting_iterator<int> lCounterEnd{static_cast<int>(aStartIndex + lSize)};

    const auto lBeginIter = thrust::make_zip_iterator(
        make_tuple(lCounterBegin, mDeviceBuffer.begin() + DEFAULT_BUFFER_OPEN_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_CLOSE_POS + aStartIndex,
                   mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + aStartIndex,
                   mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + aStartIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_AMOUNT_POS + aStartIndex));

    const auto lEndIter =
        thrust::make_zip_iterator(make_tuple(lCounterEnd, mDeviceBuffer.begin() + DEFAULT_BUFFER_OPEN_POS + lEndIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_CLOSE_POS + lEndIndex,
                                             mDeviceBuffer.begin() + DEFAULT_BUFFER_LOW_POS + lEndIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_HIGH_POS + lEndIndex,
                                             mDeviceBuffer.begin() + DEFAULT_BUFFER_VOLUME_POS + lEndIndex, mDeviceBuffer.begin() + DEFAULT_BUFFER_AMOUNT_POS + lEndIndex));

    mDeviceTempBuffer.clear();

    thrust::transform(lBeginIter, lEndIter, mDeviceTempBuffer.begin(),
                      [sx = aScaleX, tx = aTransX, sy = aScaleY, ty = aTransY, sz = aScaleZ, tz = aTransZ] __device__(const IntFloat6& a) {
                          return thrust::make_tuple(a.get<0>() * sx + tx, a.get<1>() * sy + ty, a.get<2>() * sy + ty, a.get<3>() * sy + ty, a.get<4>() * sy + ty,
                                                    a.get<5>() * sz + tz, a.get<6>() * sz + tz);
                      });

    mHostTempBuffer.assign(mDeviceTempBuffer.begin(), mDeviceTempBuffer.begin() + lSize);

    return std::make_pair(thrust::make_zip_iterator(thrust::make_tuple(mDateBuffer.begin() + aStartIndex, mHostTempBuffer.begin())),
                          thrust::make_zip_iterator(thrust::make_tuple(mDateBuffer.begin() + lEndIndex, mHostTempBuffer.end())));
}



}    // namespace abollo
